#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
    Parallel reduction kernels
*/

#ifndef _REDUCE_KERNEL_H_
#define _REDUCE_KERNEL_H_

#include <stdio.h>
#include <assert.h>

#ifdef __DEVICE_EMULATION__
#define EMUSYNC __syncthreads()
#else
#define EMUSYNC
#endif

static texture<float, 2, hipReadModeElementType> tex_ref_1;
static texture<float, 2, hipReadModeElementType> tex_ref_2;

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
    __device__ inline operator       T*()
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }

    __device__ inline operator const T*() const
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }
};

// specialize for double to avoid unaligned memory 
// access compile errors
template<>
struct SharedMemory<double>
{
    __device__ inline operator       double*()
    {
        extern __shared__ double __smem_d[];
        return (double*)__smem_d;
    }

    __device__ inline operator const double*() const
    {
        extern __shared__ double __smem_d[];
        return (double*)__smem_d;
    }
};

template <unsigned int blockSize>
__device__ void
reduceBlock(volatile float *sdata, float mySum, const unsigned int tid)
{
    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }

#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
    {
        if (blockSize >=  64) { sdata[tid] = mySum = mySum + sdata[tid + 32]; EMUSYNC; }
        if (blockSize >=  32) { sdata[tid] = mySum = mySum + sdata[tid + 16]; EMUSYNC; }
        if (blockSize >=  16) { sdata[tid] = mySum = mySum + sdata[tid +  8]; EMUSYNC; }
        if (blockSize >=   8) { sdata[tid] = mySum = mySum + sdata[tid +  4]; EMUSYNC; }
        if (blockSize >=   4) { sdata[tid] = mySum = mySum + sdata[tid +  2]; EMUSYNC; }
        if (blockSize >=   2) { sdata[tid] = mySum = mySum + sdata[tid +  1]; EMUSYNC; }
    }
}

template<bool square>
__device__ void
chamfer_and_reduce_dispatch(float *g_idata_1, float *g_idata_2, float *g_odata, unsigned int n)
{
    float *sdata = SharedMemory<float>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    const unsigned int blockSize = 256;
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    float mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    { 
        if(square)
        {
            float result = g_idata_1[i] * g_idata_2[i];
            mySum += result * result;
        }
        else
        {
            mySum += g_idata_1[i] * g_idata_2[i];
        }

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (i + blockSize < n) 
        {
            if(square)
            {
                float result = g_idata_1[i+blockSize] * g_idata_2[i+blockSize];
                result *= result;
                mySum += result;
            }
            else
            {
                mySum += g_idata_1[i+blockSize] * g_idata_2[i+blockSize];  
            }
        }
        i += gridSize;
    } 

    reduceBlock<blockSize>(sdata, mySum, tid);

    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}


extern "C" __global__ void
chamfer_and_reduce(float *g_idata_1, float *g_idata_2, float *g_odata, unsigned int n)
{
    chamfer_and_reduce_dispatch<false>(g_idata_1, g_idata_2, g_odata, n);
}

extern "C" __global__ void
squared_chamfer_and_reduce(float *g_idata_1, float *g_idata_2, float *g_odata, unsigned int n)
{
    chamfer_and_reduce_dispatch<true>(g_idata_1, g_idata_2, g_odata, n);
}


/*
    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory. 
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.  
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T_in, unsigned int blockSize, bool nIsPow2>
__device__ void
reduce(T_in *g_idata, float *g_odata, unsigned int n)
{
    float *sdata = SharedMemory<float>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    float mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        mySum += g_idata[i];
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
            mySum += g_idata[i+blockSize];  
        i += gridSize;
    } 

    reduceBlock<blockSize>(sdata, mySum, tid);

    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

/**
  Convert a linear index to a 2D texture coordinate
  */
__device__ inline void index_to_texcoord(unsigned int i, float* u, float* v, unsigned int stride)
{
    // TODO: Test with width and height as stride
    // TODO: Test with u and v swapped
        // no affect

    // TODO: implement as z-order curve
    // TODO: test order of z-order curve
        // this is too expensive to make useful

    // TODO: try to index nearby
        // can get time down from 11s to 8s, but the indexing isn't right (didn't try to get it right)
    *u = i % stride;
    *v = i / stride;
}

//template <class T_in, unsigned int blockSize, bool nIsPow2> // T_in == float always
/**
 * @param n The width * height of the texture.
 * @param stride Length or Width of texture.  which dimension is used will determine how texture lookups occur; swapping may improve cache locality
 */
template <unsigned int blockSize, bool nIsPow2>
__device__ void
tex_reduce(float *g_odata, unsigned int n, unsigned int stride)
{
    float *sdata = SharedMemory<float>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    float u, v;

    unsigned int gridSize = blockSize*2*gridDim.x;
    
    float mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        index_to_texcoord(i, &u, &v, stride);
        mySum += tex2D(tex_ref_1, u, v);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
        {
            index_to_texcoord(i+blockSize, &u, &v, stride);
            mySum += tex2D(tex_ref_1, u, v);
        }
        i += gridSize;
    } 

    reduceBlock<blockSize>(sdata, mySum, tid);

    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

//template <class T_in, unsigned int blockSize, bool nIsPow2> // T_in == float always
/**
 * Multiply a two textures and then reduce.  This will give the unnormalized chamfer distance
 * if one texture is a distance map and the other is an edge mask containing 1.0's and 0.0's.  To get the normalization factor, pass the edge mask texture to tex_count.
 *
 * @param n The width * height of the texture.
 * @param stride Length or Width of texture.  which dimension is used will determine how texture lookups occur; swapping may improve cache locality
 */
template <unsigned int blockSize, bool nIsPow2>
__device__ void
chamfer_reduce(float *g_odata, unsigned int n, unsigned int stride)
{
    float *sdata = SharedMemory<float>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    float u, v;

    unsigned int gridSize = blockSize*2*gridDim.x;
    
    float mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        index_to_texcoord(i, &u, &v, stride);
        mySum += tex2D(tex_ref_1, u, v) * tex2D(tex_ref_2, u, v);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
        {
            index_to_texcoord(i+blockSize, &u, &v, stride);
            mySum += tex2D(tex_ref_1, u, v) * tex2D(tex_ref_2, u, v);
        }
        i += gridSize;
    } 

    reduceBlock<blockSize>(sdata, mySum, tid);

    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

//template <class T_in, unsigned int blockSize, bool nIsPow2> // T_in == float always
/**
 * Count the number of non-zero elements in a texture.
 * 
 * @param n The width * height of the texture.
 * @param stride Length or Width of texture.  which dimension is used will determine how texture lookups occur; swapping may improve cache locality
 */
template <unsigned int blockSize, bool nIsPow2>
__device__ void
tex_count(float *g_odata, unsigned int n, unsigned int stride)
{
    float *sdata = SharedMemory<float>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    float u, v;

    unsigned int gridSize = blockSize*2*gridDim.x;
    
    float mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        index_to_texcoord(i, &u, &v, stride);
        if(tex2D(tex_ref_1, u, v) != 0.0)
            mySum += 1;;

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
        {
            index_to_texcoord(i+blockSize, &u, &v, stride);
            if(tex2D(tex_ref_1, u, v) != 0.0)
                mySum += 1;
        }

        i += gridSize;
    } 

    reduceBlock<blockSize>(sdata, mySum, tid);

    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}


/*
 * Since we can't call template kernels from the Cuda Driver API,
 * we wrap them here as C functions, then call those through the Driver.
 */

/*********************************
 * The 'float' versions.
 *********************************/

extern "C" __global__ void reduce_float_1_true(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 1, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_2_true(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 2, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_4_true(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 4, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_8_true(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 8, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_16_true(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 16, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_32_true(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 32, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_64_true(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 64, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_128_true(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 128, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_256_true(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 256, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_512_true(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 512, true>(g_idata, g_odata, n);
}


extern "C" __global__ void reduce_float_1_false(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 1, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_2_false(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 2, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_4_false(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 4, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_8_false(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 8, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_16_false(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 16, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_32_false(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 32, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_64_false(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 64, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_128_false(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 128, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_256_false(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 256, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_float_512_false(float *g_idata, float *g_odata, unsigned int n)
{
    reduce<float, 512, false>(g_idata, g_odata, n);
}

/******************
  Texture version (experimental)
*********************/

extern "C" __global__ void tex_reduce_256_false(float *g_odata, unsigned int n, unsigned int stride)
{
    tex_reduce<256, false>(g_odata, n, stride);
}

extern "C" __global__ void tex_count_256_false(float *g_odata, unsigned int n, unsigned int stride)
{
    tex_count<256, false>(g_odata, n, stride);
}

extern "C" __global__ void chamfer_reduce_256_false(float *g_odata, unsigned int n, unsigned int stride)
{
    chamfer_reduce<256, false>(g_odata, n, stride);
}

/*********************************
 * The 'uchar' versions.
 *
 * These receive an array of unsigned chars, but 
 * return an array of ints, since one byte will
 * overflow too quickly.
 *********************************/

extern "C" __global__ void reduce_uchar_1_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 1, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_2_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 2, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_4_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 4, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_8_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 8, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_16_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 16, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_32_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 32, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_64_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 64, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_128_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 128, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_256_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 256, true>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_512_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 512, true>(g_idata, g_odata, n);
}


extern "C" __global__ void reduce_uchar_1_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 1, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_2_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 2, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_4_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 4, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_8_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 8, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_16_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 16, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_32_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 32, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_64_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 64, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_128_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 128, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_256_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 256, false>(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_uchar_512_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    reduce<unsigned char, 512, false>(g_idata, g_odata, n);
}


/* **********************************************************
   PACKED_FLOAT_REDUCE()

   This version of reduce() is for arrays of float values packed
   into RGBA colors.  This is the case when an RGBA texture is used
   to store the float values.  It tries to handle the case where the CPU
   and GPU have different endian-ness by swapping the bytes, but
   that branch of the code is currently untested, so proceed with caution.

 **********************************************************/


/**
* See the other reduce() kernel for usage notes.

* @tparam blockSize The number of threads per block.  Valid values are 2^m where m is between 0 and 9.
* @tparam nIsPow2 Set this to true if n is a power of 2.  The code is optimized for this case.
* @tparam odd_endian_parity Set this to true when the endian-ness of the CPU and GPU differ. (UNTESTED)
*
* @param g_idata Input data.  The array of bytes representing the float values.  Every chunk of 4 bytes stores a float value.
* @param g_odata Output data.  Result is output as a float.
* @param n the number of float elements in g_idata.  Note that the number of byte elements in g_idata is 4*n.
*/
template <unsigned int blockSize, bool nIsPow2, bool odd_endian_parity>
__device__ void
packed_float_reduce(unsigned char *g_idata, float *g_odata, unsigned int n)
{
    float *sdata = SharedMemory<float>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    float mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        unsigned char address = i * 4; // could compute this when i is created to minimize register usage

        if(odd_endian_parity)
        {
            // UNTESTED!!!  50% chance that this will give the wrong answer!
            // also, it may be easier to handle this in host code, by passing GL_UINT_8_8_8_8_REV
            // to the glReadPixels call for the PBO.
                union {
                    unsigned char bytes[4];
                    float value;
                } pack;
                pack.bytes[3] = g_idata[address];
                pack.bytes[2] = g_idata[address + 1];
                pack.bytes[1] = g_idata[address + 2];
                pack.bytes[0] = g_idata[address + 3];
                mySum = pack.value;
        }
        else
        {
            mySum += *(float*)(&g_idata[address]);
        }


        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
        {
            address = (i + blockSize) * 4;
            if(odd_endian_parity)
            {
                union {
                    unsigned char bytes[4];
                    float value;
                } pack;
                pack.bytes[3] = g_idata[address];
                pack.bytes[2] = g_idata[address + 1];
                pack.bytes[1] = g_idata[address + 2];
                pack.bytes[0] = g_idata[address + 3];
                mySum = pack.value;
            }
            else
            {
                mySum += *(float*)(&g_idata[address]);
            }
        }
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }
    
#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile float* smem = sdata;
        if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; EMUSYNC; }
        if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; EMUSYNC; }
        if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; EMUSYNC; }
        if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; EMUSYNC; }
        if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; EMUSYNC; }
        if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; EMUSYNC; }
    }
    
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

extern "C" __global__ void packed_float_reduce_1_false_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<1, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_1_false_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<1, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_1_true_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<1, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_1_true_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<1, true, true>(g_idata, g_odata, n); }

extern "C" __global__ void packed_float_reduce_2_false_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<2, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_2_false_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<2, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_2_true_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<2, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_2_true_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<2, true, true>(g_idata, g_odata, n); }

extern "C" __global__ void packed_float_reduce_4_false_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<4, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_4_false_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<4, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_4_true_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<4, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_4_true_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<4, true, true>(g_idata, g_odata, n); }

extern "C" __global__ void packed_float_reduce_8_false_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<8, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_8_false_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<8, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_8_true_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<8, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_8_true_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<8, true, true>(g_idata, g_odata, n); }

extern "C" __global__ void packed_float_reduce_16_false_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<16, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_16_false_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<16, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_16_true_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<16, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_16_true_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<16, true, true>(g_idata, g_odata, n); }

extern "C" __global__ void packed_float_reduce_32_false_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<32, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_32_false_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<32, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_32_true_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<32, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_32_true_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<32, true, true>(g_idata, g_odata, n); }

extern "C" __global__ void packed_float_reduce_64_false_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<64, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_64_false_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<64, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_64_true_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<64, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_64_true_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<64, true, true>(g_idata, g_odata, n); }

extern "C" __global__ void packed_float_reduce_128_false_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<128, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_128_false_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<128, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_128_true_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<128, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_128_true_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<128, true, true>(g_idata, g_odata, n); }

extern "C" __global__ void packed_float_reduce_256_false_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<256, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_256_false_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<256, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_256_true_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<256, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_256_true_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<256, true, true>(g_idata, g_odata, n); }

extern "C" __global__ void packed_float_reduce_512_false_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<512, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_512_false_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<512, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_512_true_false(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<512, true, true>(g_idata, g_odata, n); }
extern "C" __global__ void packed_float_reduce_512_true_true(unsigned char *g_idata, float *g_odata, unsigned int n)
{ packed_float_reduce<512, true, true>(g_idata, g_odata, n); }




extern "C"
bool isPow2(unsigned int x);


/*
////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T>
void 
cuda_reduce_host(int size, int threads, int blocks, 
       T *d_idata, T *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // someday more threads may be available, but this function will have to be
    // expanded to accomodate
    assert(threads <= 512);

    // TODO check that size is less than or equal to the devices max
    // TODO check that blocks are less than or equal to the device max

    // when there is only one warp per block, we need to allocate two warps 
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

    if (isPow2(size))
    {
        switch (threads)
        {
        case 512:
            reduce<T, 512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 256:
            reduce<T, 256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 128:
            reduce<T, 128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 64:
            reduce<T,  64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 32:
            reduce<T,  32, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 16:
            reduce<T,  16, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  8:
            reduce<T,   8, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  4:
            reduce<T,   4, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  2:
            reduce<T,   2, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  1:
            reduce<T,   1, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        }
    }
    else
    {
        switch (threads)
        {
        case 512:
            reduce<T, 512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 256:
            reduce<T, 256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 128:
            reduce<T, 128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 64:
            reduce<T,  64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 32:
            reduce<T,  32, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 16:
            reduce<T,  16, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  8:
            reduce<T,   8, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  4:
            reduce<T,   4, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  2:
            reduce<T,   2, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  1:
            reduce<T,   1, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        }
    }
}

// Instantiate the reduction function for 3 types
template void 
cuda_reduce_host<int>(int size, int threads, int blocks, 
            int *d_idata, int *d_odata);

template void 
cuda_reduce_host<float>(int size, int threads, int blocks, 
              float *d_idata, float *d_odata);

template void 
cuda_reduce_host<double>(int size, int threads, int blocks, 
               double *d_idata, double *d_odata);
*/

#endif // #ifndef _REDUCE_KERNEL_H_
